#include "hip/hip_runtime.h"
#include <functional>
#include <stdio.h>
#include "mkl.h"

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
               hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e]
                * B.elements[e * B.width + col];
    C.elements[row * C.width + col] = Cvalue;
}

auto init_A = [](int i, int j)->float { return i == j ? 1 : 0; };
auto init_B = [](int i, int j)->float { return j - i > 0 ? j - i : 0; };
auto init_zeros = [](int i, int j)->float { return 0; };
void init_mat(Matrix mat, std::function<float (int, int)> initializer)
{
	for(int i = 0; i < mat.height; i++)
	{
		for(int j = 0; j < mat.width; j++)
		{
			mat.elements[j + i * mat.width] = initializer(i, j);
		}
	}
}
void print_mat(Matrix M) {
	for(int i = 0; i < M.height; i++)
	{
		for(int j = 0; j < M.width; j++)
		{
			printf("%d ", (int)M.elements[j + i * M.width]);
		}
		printf("\n");
	}
}

void ref_dgemm(Matrix A, Matrix B, Matrix C)
{
    int m = A.height;
    int n = B.width;
    int k = A.width;
    float alpha = 1.0;
    float beta = 0.0;
    cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans,
                m, n, k, alpha, A.elements, k, B.elements, n, beta, C.elements, n);
}

float compare_mat(Matrix A, Matrix Ref)
{
    float error = 0.0;
    int idx;
    for(int i = 0; i < Ref.height; i++)
	{
		for(int j = 0; j < Ref.width; j++)
		{
            idx = j + i * Ref.width;
			error += Ref.elements[idx] - A.elements[idx];
		}
	}
    return error;
}

int main(int argc, char ** argv)
{
    Matrix A, B, C;
	A.height = 32;
	A.width = 32;
	B.width = 32;
	B.height = A.width;
	C.height = A.height;
	C.width = B.width;
	A.elements = (float *) mkl_malloc(A.width * A.height * sizeof(float), 64);
	B.elements = (float *) mkl_malloc(B.width * B.height * sizeof(float), 64);
	C.elements = (float *) mkl_malloc(C.width * C.height * sizeof(float), 64);
	init_mat(A, init_A);
	init_mat(B, init_B);
	init_mat(C, init_zeros);
    MatMul(A, B, C);

    Matrix ref_C;
    ref_C.height = C.height;
    ref_C.width = C.width;
	ref_C.elements = (float *) mkl_malloc(C.width * C.height * sizeof(float), 64);
	init_mat(ref_C, init_zeros);
    ref_dgemm(A, B, ref_C);

    print_mat(C);

    float tolerance = 0.0;
    float error = compare_mat(C, ref_C);
    if(error > tolerance)
    {
        printf("FAIL: error = %f\n", error);
    }

    // print_mat(ref_C);

	mkl_free(A.elements);
	mkl_free(B.elements);
	mkl_free(C.elements);
	mkl_free(ref_C.elements);
}
