#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.


  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

  // 0 - cheat and copy to host for sanity
  unsigned int channelSize = numRows * numCols;
  float* h_logLuminance = (float*)malloc(channelSize * sizeof(float));
  checkCudaErrors(hipMemcpy(h_logLuminance,   d_logLuminance,   channelSize * sizeof(float), hipMemcpyDeviceToHost));

  // 1 min/max luminance
  min_logLum = h_logLuminance[0];
  max_logLum = h_logLuminance[0];
  for (size_t i = 1; i < numCols * numRows; ++i) {
    min_logLum = std::min(h_logLuminance[i], min_logLum);
    max_logLum = std::max(h_logLuminance[i], max_logLum);
  }

  // 2 range
  float range = max_logLum - min_logLum;
  printf("max %f, min %f, range %f\n", max_logLum, min_logLum, range);

  // 3 histogram
  unsigned int* h_bins = (unsigned int*)malloc(numBins * sizeof(unsigned int));
  for(int i = 0; i < numBins; i++){
    h_bins[i] = 0;
  }
  for(int i = 0; i < channelSize; i++){
    int bin = (h_logLuminance[i] - min_logLum) / range * numBins;
    h_bins[bin]++;
  }
  printf("\nIDX  \t");
  for(int i = 0; i < numBins; i++){
    printf("[%d]\t", i);
  }
  printf("\nHIST \t");
  for(int i = 0; i < numBins; i++){
    printf("%d\t", h_bins[i]);
  }

  // 4
  unsigned int* h_cdf = (unsigned int*)malloc(numBins * sizeof(unsigned int));
  h_cdf[0] = 0;
  for(int i = 1; i < numBins; i++){
    h_cdf[i] = h_cdf[i-1] + h_bins[i-1];
  }

  printf("\nCDF  \t");
  for(int i = 0; i < numBins; i++){
    printf("%d\t", h_cdf[i]);
  }
  checkCudaErrors(hipMemcpy(d_cdf,   h_cdf,   numBins*sizeof(unsigned int), hipMemcpyHostToDevice));

  free(h_logLuminance);
  free(h_bins);
  free(h_cdf);
}
